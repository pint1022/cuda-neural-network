
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <functional>
#include <stdlib.h>     /* srand, rand */
#include <time.h>       /* time */
#include <chrono>

template<typename T>
void initialize_matrix(T* M, int rows, int cols, std::function<float()> F) {
  for(int i = 0; i < rows; i++){
    for(int j = 0; j < cols; j++){
      M[i * cols + j] = F();
    }
  }
}

template<typename T>
void initialize_matrix(T* M, int rows, int cols, std::function<float(int, int)> F) {
  for(int i = 0; i < rows; i++){
    for(int j = 0; j < cols; j++){
      M[i * cols + j] = F(i, j);
    }
  }
}

template<typename T>
void print_matrix(T* M, int rows, int cols) {
  for(int i = 0; i < rows; i++){
    for(int j = 0; j < cols; j++){
        std::cout << M[i * cols + j] << " ";
    }
    std::cout << "\n";
  }
}


template<typename T>
T maxDiff(T* A1, T* A2, int rows, int cols){
  T maxDiff = A1[0] - A2[0];
  for(int i = 0; i < rows; i++) {
    for(int j = 0; j < cols; j++){
      T diff = abs(A1[i * cols + j] - A2[i * cols + j]);
      if( diff > maxDiff) {
          maxDiff = diff;
      }
    }
  }

  
  return maxDiff;
}

template<typename T>
void check_copy(T* dM, T* hM, int d_size, char* label) {
  T* cp_host = (T*) malloc(d_size*sizeof(T));

  std::cout << "\n\n" << label << std::endl;

	hipMemcpy(cp_host, dM, d_size, hipMemcpyDeviceToHost);
  std::cout << "CPU original" << std::endl;
  print_matrix<T>( hM, 10, 10);
  std::cout << "GPU copy" << std::endl;
  print_matrix<T>( cp_host, 10, 10);

  free(cp_host);
}

#define ROW_TILE_WIDTH  32
#define COL_TILE_WIDTH  32

#define EPSILON         (1e-5)

template<typename T>
__global__
void naive_matrix_multiply(T *A, T *B, T* C, int width, int cRows, int cCols)
{
  __shared__ T shATile[ROW_TILE_WIDTH][COL_TILE_WIDTH];
  __shared__ T shBTile[ROW_TILE_WIDTH][COL_TILE_WIDTH];

  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  T pValue = 0;

  // iterate for width/COL_TILE_WIDTH number of times
  // to compute the C tile
  for(int p = 0; p < width/COL_TILE_WIDTH; p++) {
    //load values to tiles from A and B
    shATile[threadIdx.y][threadIdx.x] = A[row * width + p * ROW_TILE_WIDTH + threadIdx.x];
    shBTile[threadIdx.y][threadIdx.x] = B[(p * COL_TILE_WIDTH + threadIdx.y) * cCols + col];

    // wait until all threads finish loading values
    __syncthreads();
    // update pValue for this thread
    for(int i = 0; i < COL_TILE_WIDTH;  i++) pValue += shATile[threadIdx.y][i] * shBTile[i][threadIdx.x];
    // wait until all threads finish computing pValue before overwriting
    __syncthreads();

  }
  
  C[row * cCols + col] = pValue;

}

template<typename T>
void naive_matrix_multiply_cpu(T *A, T *B, T* C, int width, int C_rows, int C_cols){
  
  for(int i = 0; i < C_rows; i++)
    for(int j = 0; j < C_cols; j++){
      T value = 0.0f;
      for(int k = 0; k < width; k++){
        value += A[i * width + k] * B[k * C_cols + j];
      }

     
      C[i * C_cols + j] = value;
    }
}

extern "C" void perform_matmul() {
  int A_rows = 1 << 8;
  int A_cols = 1 << 10;
  int B_cols = 1 << 11;

  int B_rows = A_cols;
  int C_rows = A_rows;
  int C_cols = B_cols;
  int A_size = A_rows * A_cols;
  int B_size = B_rows * B_cols;
  int C_size = C_rows * C_cols;
  float *A, *B, *C, *C_host;
  float *A_cpu, *B_cpu, *C_cpu;
  // timing
  hipEvent_t start_gpu, stop_gpu;
  float gpu_time_ms = 0;
  hipEventCreate(&start_gpu);
  hipEventCreate(&stop_gpu);
  
  std::cout << "A size: " << A_size << ", B size: " << B_size << ", C Size: " << C_size << std::endl;  
  std::cout << "A: " << A_rows << "x" << A_cols << ", B: " << B_rows << "x" << B_cols <<  ", C: " << C_rows << "x" << C_cols << std::endl;  
  // Allocate Unified Memory – accessible from CPU or GPU
  hipMalloc(&A, A_size*sizeof(float));

  hipMalloc(&B, B_size*sizeof(float));
  hipMalloc(&C, C_size*sizeof(float));
  C_host = (float*) malloc(C_size*sizeof(float));


  A_cpu = (float*) malloc(A_size*sizeof(float));
  B_cpu = (float*) malloc(B_size*sizeof(float));
  C_cpu = (float*) malloc(C_size*sizeof(float));

  // initialize A and B matrices
  auto all_ones = []() -> float {
    return 1.0f;
  };

  srand (time(NULL));
  auto rand_numbers = []() -> float {
    return static_cast<float>(rand())/(static_cast<float>(RAND_MAX/1000));
  };

  auto index_based = [](int i, int j) -> float {
    return j;
  };

  initialize_matrix<float>(A_cpu, A_rows, A_cols, rand_numbers);
	hipMemcpy(A, A_cpu, A_size * sizeof(float), hipMemcpyHostToDevice);  

  initialize_matrix<float>(B_cpu, B_rows, B_cols, rand_numbers);
	hipMemcpy(B, B_cpu, B_size * sizeof(float), hipMemcpyHostToDevice);


  // launch kernel

  dim3 dim_grid(C_cols/COL_TILE_WIDTH, C_rows/ROW_TILE_WIDTH, 1);
  dim3 dim_block(COL_TILE_WIDTH, ROW_TILE_WIDTH, 1);

  hipEventRecord(start_gpu);
  naive_matrix_multiply<float><<<dim_grid, dim_block>>>(A, B, C, A_cols, C_rows, C_cols);
  hipEventRecord(stop_gpu);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();
	hipMemcpy(C_host, C, C_size * sizeof(float), hipMemcpyDeviceToHost);

  hipEventSynchronize(stop_gpu);
  hipEventElapsedTime(&gpu_time_ms, start_gpu, stop_gpu);
  

  // check results on CPU
  auto t1 = std::chrono::system_clock::now();
  naive_matrix_multiply_cpu<float>(A_cpu, B_cpu, C_cpu, A_cols, C_rows, C_cols);
  auto t2 = std::chrono::system_clock::now();

  if(fabsf(maxDiff<float>(C_host, C_cpu, C_rows, C_cols)) <= (float)EPSILON )
     std::cout << "PASS" << std::endl;
  else {
     std::cout << "FAIL" << std::endl;
     std::cout << "GPU result [0:9, 0:9]" << std::endl;
     print_matrix<float>( C_host, 10, 10);
     std::cout << "CPU result [0:9, 0:9]" << std::endl;
     print_matrix<float>( C_cpu, 10, 10);

  }

  auto cpu_time_ms = std::chrono::duration_cast<std::chrono::microseconds>(t2 - t1).count()/1000.0f;
  std::cout << "GPU time = " << gpu_time_ms << "ms" << std::endl;
  std::cout << "CPU time = " << cpu_time_ms << "ms" << std::endl;
  std::cout << "Speedup = " << cpu_time_ms/gpu_time_ms << std::endl;
  
  // Free memory
  hipFree(A);
  hipFree(B);
  hipFree(C);
  free(C_host);

  free(A_cpu);
  free(B_cpu);
  free(C_cpu);  
}