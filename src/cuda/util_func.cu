#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <functional>
#include <stdlib.h>     /* srand, rand */
#include <time.h>       /* time */
#include <chrono>
#include "cublas_func.cuh"

template<typename T>
void initialize_matrix(T* M, int rows, int cols, std::function<double()> F) {
  for(int i = 0; i < rows; i++){
    for(int j = 0; j < cols; j++){
      M[i * cols + j] = F();
    }
  }
}

template<typename T>
void initialize_matrix(T* M, int rows, int cols, std::function<double(int, int)> F) {
  for(int i = 0; i < rows; i++){
    for(int j = 0; j < cols; j++){
      M[i * cols + j] = F(i, j);
    }
  }
}

template<typename T>
void print_matrix(T* M, int rows, int cols) {
  for(int i = 0; i < rows; i++){
    for(int j = 0; j < cols; j++){
        std::cout << M[i * cols + j] << " ";
    }
    std::cout << "\n";
  }
}


template<typename T>
T maxDiff(T* A1, T* A2, int rows, int cols){
  T maxDiff = A1[0] - A2[0];
  for(int i = 0; i < rows; i++) {
    for(int j = 0; j < cols; j++){
      T diff = abs(A1[i * cols + j] - A2[i * cols + j]);
      if( diff > maxDiff) {
          maxDiff = diff;
      }
    }
  }

  
  return maxDiff;
}

template<typename T>
void check_copy(T* dM, T* hM, int d_size, char* label) {
  T* cp_host = (T*) malloc(d_size*sizeof(T));

  std::cout << "\n\n" << label << std::endl;

	hipMemcpy(cp_host, dM, d_size, hipMemcpyDeviceToHost);
  std::cout << "CPU original" << std::endl;
  print_matrix<T>( hM, 10, 10);
  std::cout << "GPU copy" << std::endl;
  print_matrix<T>( cp_host, 10, 10);

  free(cp_host);
}

#define ROW_TILE_WIDTH  32
#define COL_TILE_WIDTH  32

#define EPSILON         (1e-5)

template<typename T>
__global__
void naive_matrix_multiply(T *A, T *B, T* C, int width, int cRows, int cCols)
{
  __shared__ T shATile[ROW_TILE_WIDTH][COL_TILE_WIDTH];
  __shared__ T shBTile[ROW_TILE_WIDTH][COL_TILE_WIDTH];

  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  T pValue = 0;

  // iterate for width/COL_TILE_WIDTH number of times
  // to compute the C tile
  for(int p = 0; p < width/COL_TILE_WIDTH; p++) {
    //load values to tiles from A and B
    shATile[threadIdx.y][threadIdx.x] = A[row * width + p * ROW_TILE_WIDTH + threadIdx.x];
    shBTile[threadIdx.y][threadIdx.x] = B[(p * COL_TILE_WIDTH + threadIdx.y) * cCols + col];

    // wait until all threads finish loading values
    __syncthreads();
    // update pValue for this thread
    for(int i = 0; i < COL_TILE_WIDTH;  i++) pValue += shATile[threadIdx.y][i] * shBTile[i][threadIdx.x];
    // wait until all threads finish computing pValue before overwriting
    __syncthreads();

  }
  
  C[row * cCols + col] = pValue;

}

template<typename T>
void naive_matrix_multiply_cpu(T *A, T *B, T* C, int width, int C_rows, int C_cols){
  
  for(int i = 0; i < C_rows; i++)
    for(int j = 0; j < C_cols; j++){
      T value = 0.0f;
      for(int k = 0; k < width; k++){
        value += A[i * width + k] * B[k * C_cols + j];
      }

     
      C[i * C_cols + j] = value;
    }
}

extern "C" void perform_matmul(double* A_cpu, double *B_cpu, double *C_host, int a_row, int a_col, int b_row, int b_col, int flag) {
  // int A_rows = 1 << 8;
  // int A_cols = 1 << 10;
  // int B_cols = 1 << 11;

  // int B_rows = A_cols;
  // int C_rows = A_rows;
  // int C_cols = B_cols;

  int A_rows = a_row;
  int A_cols = a_col;
  int B_cols = b_col;

  int B_rows = A_cols;
  int C_rows = A_rows;
  int C_cols = B_cols;

  int A_size = A_rows * A_cols;
  int B_size = B_rows * B_cols;
  int C_size = C_rows * C_cols;
  double *A, *B, *C;
  // double *A_cpu, *B_cpu, *C_cpu;
  double *C_cpu;
  // timing
  hipEvent_t start_gpu, stop_gpu;
  float gpu_time_ms = 0;
  hipEventCreate(&start_gpu);
  hipEventCreate(&stop_gpu);
  
  std::cout << "A size: " << A_size << ", B size: " << B_size << ", C Size: " << C_size << std::endl;  
  std::cout << "A: " << A_rows << "x" << A_cols << ", B: " << B_rows << "x" << B_cols <<  ", C: " << C_rows << "x" << C_cols << std::endl;  
  // Allocate Unified Memory – accessible from CPU or GPU
  hipMalloc(&A, A_size*sizeof(double));

  hipMalloc(&B, B_size*sizeof(double));
  hipMalloc(&C, C_size*sizeof(double));
  // C_host = (double*) malloc(C_size*sizeof(double));


  // A_cpu = (double*) malloc(A_size*sizeof(double));
  // B_cpu = (double*) malloc(B_size*sizeof(double));
  C_cpu = (double*) malloc(C_size*sizeof(double));

  // initialize A and B matrices
  auto all_ones = []() -> double {
    return 1.0f;
  };

  srand (time(NULL));
  auto rand_numbers = []() -> double {
    return static_cast<double>(rand())/(static_cast<double>(RAND_MAX/1000));
  };

  auto index_based = [](int i, int j) -> double {
    return j;
  };

  // initialize_matrix<double>(A_cpu, A_rows, A_cols, rand_numbers);
	hipMemcpy(A, A_cpu, A_size * sizeof(double), hipMemcpyHostToDevice); 
  
  if(const char* env_p = std::getenv("ALNAIR_DBG")) {
    // std::cout << "Debug mode: " << env_p << '\n';
    if (strlen(env_p) > 0)
        check_copy<double>(A, A_cpu, A_size, "A matrix");
  } 

  // initialize_matrix<double>(B_cpu, B_rows, B_cols, rand_numbers);
	hipMemcpy(B, B_cpu, B_size * sizeof(double), hipMemcpyHostToDevice);

  // launch kernel

  char kernel_name[20];
  if (flag == 1) {
    strcpy(kernel_name, "cublas");

    hipEventRecord(start_gpu);
    gpu_blas_mmul((const float*) A_cpu, (const float *) B_cpu, (float *)C,  A_cols, C_rows, C_cols);
    hipEventRecord(stop_gpu);
  } else {
    strcpy(kernel_name, "tiled");

    dim3 dim_grid(C_cols/COL_TILE_WIDTH, C_rows/ROW_TILE_WIDTH, 1);
    dim3 dim_block(COL_TILE_WIDTH, ROW_TILE_WIDTH, 1);

    naive_matrix_multiply<double><<<dim_grid, dim_block>>>(A, B, C, A_cols, C_rows, C_cols);
    hipEventRecord(stop_gpu);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();
  }
  hipMemcpy(C_host, C, C_size * sizeof(double), hipMemcpyDeviceToHost);

  hipEventSynchronize(stop_gpu);
  hipEventElapsedTime(&gpu_time_ms, start_gpu, stop_gpu);

  

  // check results on CPU
  auto t1 = std::chrono::system_clock::now();
  naive_matrix_multiply_cpu<double>(A_cpu, B_cpu, C_cpu, A_cols, C_rows, C_cols);
  auto t2 = std::chrono::system_clock::now();

  if(fabsf(maxDiff<double>(C_host, C_cpu, C_rows, C_cols)) <= (double)EPSILON )
     std::cout << "PASS" << std::endl;
  else {
     std::cout << "FAIL" << std::endl;
     std::cout << "GPU result [0:9, 0:9]" << std::endl;
     print_matrix<double>( C_host, 10, 10);
     std::cout << "CPU result [0:9, 0:9]" << std::endl;
     print_matrix<double>( C_cpu, 10, 10);

  }

  auto cpu_time_ms = std::chrono::duration_cast<std::chrono::microseconds>(t2 - t1).count()/1000.0f;
  std::cout << kernel_name << " GPU time = " << gpu_time_ms << "ms" << std::endl;
  std::cout << "CPU time = " << cpu_time_ms << "ms" << std::endl;
  std::cout << "Speedup = " << cpu_time_ms/gpu_time_ms << std::endl;
  
  // Free memory
  hipFree(A);
  hipFree(B);
  hipFree(C);
  // free(C_host);

  // free(A_cpu);
  // free(B_cpu);
  free(C_cpu);  
}