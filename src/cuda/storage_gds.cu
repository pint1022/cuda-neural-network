#include "hip/hip_runtime.h"
﻿#include <storage_gds.cuh>
#include <utils.cuh>

#include <hiprand/hiprand_kernel.h>
#include <>

#include <cmath>

GDSStorage::~GDSStorage()
{
    if (this->gpu)
        hipFree(this->device_data);
    this->gpu = false;
    std::cout << "\n GDSStorageDestructor executed";
}

GDSStorage::GDSStorage(const std::vector<int> &_shape) : shape(_shape), flag(HOST_MEM) {
  int size = 1;
  for (int i = 0; i < _shape.size(); i++) {
    size *= _shape[i];
  }

  if (flag==GDS_MEM) {
    if (this->gpu) {
        std::cout << "Already registered?" << std::endl;
      	cuFileBufDeregister((char*)this->device_data);
        hipFree(this->device_data);
    } 
	  hipMalloc(&(this->device_data), size);
    cuFileBufRegister((char*)this->device_data, size, 0);
    this->gpu = true;
  }
  else 
      std::cout << "Not supported" << std::endl;
}

GDSStorage::GDSStorage(const std::vector<int> &_shape, float value) : shape(_shape) , flag(HOST_MEM){
  int size = 1;
  for (int i = 0; i < _shape.size(); i++) {
    size *= _shape[i];
  }

  if (flag==GDS_MEM) {
    if (this->gpu) {
        std::cout << "Already registered?" << std::endl;
      	cuFileBufDeregister((char*)this->device_data);
        hipFree(this->device_data);
    } 
	  hipMalloc(&(this->device_data), size);
    cuFileBufRegister((char*)this->device_data, size, 0);
    this->gpu = true;
  }
  else 
      std::cout << "Not supported" << std::endl;
}

GDSStorage::GDSStorage(const std::vector<int> &_shape,
                 const std::vector<float> &_data)
    : shape(_shape),  flag(HOST_MEM){
  this->check_size();
}

GDSStorage::GDSStorage(const GDSStorage &other) { *this = other; }

GDSStorage &GDSStorage::operator=(const GDSStorage &other) {
  // if (this != &other) {
  //   this->shape = other.shape;
  //   this->data = other.data;
  // }

  // return *this;
  std::cout << "not supported" << std::endl;
}

GDSStorage::GDSStorage(const std::vector<int> &_shape, int flag) : shape(_shape) {
  int size = 1;
  for (int i = 0; i < _shape.size(); i++) {
    size *= _shape[i];
  }

  if (flag==GDS_MEM) {
    if (this->gpu) {
        std::cout << "Already registered?" << std::endl;
      	cuFileBufDeregister((char*)this->device_data);
        hipFree(this->device_data);
    } 
	  hipMalloc(&(this->device_data), size);
    cuFileBufRegister((char*)this->device_data, size, 0);
    this->gpu = true;
  }
  else 
    printf("Not implemented\n");
}

GDSStorage::GDSStorage(GDSStorage &&other) 
{ 
  if (flag==GDS_MEM)
    printf("Not implemented\n");
  else 
    *this = std::move(other); 
}

GDSStorage &GDSStorage::operator=(GDSStorage &&other) {
  if (this != &other) {
    this->shape = std::move(other.shape);
    if (flag == GDS_MEM)
        // this->device_data = std::move(other.device_data);
        printf("Not implemented\n");
    else 
        printf("Not implemented\n");
        // this->data = std::move(other.data);
  }
  return *this;
}

void GDSStorage::reshape(const std::vector<int> &_shape) {
  this->shape = _shape;
  this->check_size();
}

void GDSStorage::resize(const std::vector<int> &_shape) {

    if (flag == GDS_MEM) {
        // this->device_data = std::move(other.device_data);
        if (this->gpu) {
            std::cout << "Already registered?" << std::endl;
            cuFileBufDeregister((char*)this->device_data);
            hipFree(this->device_data);
        } 
        this->shape = _shape;

        int size = 1;
        for (int i = 0; i < _shape.size(); i++) {
            size *= _shape[i];
        }
        hipMalloc(&(this->device_data), size);
        cuFileBufRegister((char*)this->device_data, size, 0);
        this->gpu = true;           
    }
    else {
        std::cout << "Not supported" << std::endl;
        // this->shape = _shape;

        // int size = 1;
        // for (int i = 0; i < _shape.size(); i++) {
        //     size *= _shape[i];
        // }

        // if (size != this->data.size()) {
        //     this->data.resize(size);
        // }
    }
}

__global__ void storage_xavier(float *a, int size, float scale,
                               hiprandState *cs) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size) {
    hiprand_init(1234, index, 0, &cs[index]);
    a[index] = (hiprand_uniform(&cs[index]) * 2 - 1) * scale;
  }
}

void GDSStorage::xavier(size_t in_size, size_t out_size) {
  // float *a_ptr = RAW_PTR(this->data);
  float *a_ptr = (float *)this->device_data;
  int size;
  if (flag == HOST_MEM)
        std::cout << "Not supported" << std::endl;
  else
      size = sizeof(this->device_data);

  int grid_size = ceil((float)(size) / BLOCK_SIZE);

  thrust::device_vector<hiprandState> cs(size);
  hiprandState *cs_ptr = RAW_PTR(cs);
  float scale = std::sqrt((float)6) / std::sqrt((float)(in_size) + out_size);
  storage_xavier<<<grid_size, BLOCK_SIZE>>>(a_ptr, size, scale, cs_ptr);

  CUDA_POST_KERNEL_CHECK;
}

void GDSStorage::check_size() {
  int size = 1;
  for (int i = 0; i < this->shape.size(); i++) {
    size *= this->shape[i];
  }
  if (flag == HOST_MEM)
        std::cout << "Not supported" << std::endl;
  else
      CHECK_EQ(size, sizeof(this->device_data), "GDSStorage: size error");

}