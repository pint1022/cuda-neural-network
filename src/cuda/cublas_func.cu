
#include <hipblas.h>
#include <hip/hip_runtime.h>

// Multiply the arrays A and B on GPU and save the result in C
// C(m,n) = A(m,k) * B(k,n)
hipblasStatus_t  gpu_blas_mmul(const hipblasHandle_t * handle, const float *A, const float *B, float *C, const int m, const int k, const int n) {
    int lda=m,ldb=k,ldc=m;
    const float alf = 1;
    const float bet = 0;
    const float *alpha = &alf;
    const float *beta = &bet;

    // Create a handle for CUBLAS
    // cublasHandle_t handle;
    // checkCublas(cublasCreate(&handle));

    // Do the actual multiplication
    return (hipblasSgemm(*handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc));

    // Destroy the handle
    // cublasDestroy(handle);
}